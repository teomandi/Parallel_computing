#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <timer.h>

#include "convolution.h"

float convolution(float ***global, int dimension_x, int dimension_y, float filter[FILTER_SIZE][FILTER_SIZE], int *rep){

	int i, j, size, stop=1;
	float **grid_gpu, **new_grid_gpu, **new_grid;

	size = dimension_y * dimension_x * sizeof(float);

	Create_Grid(dimension_x, dimension_y,&new_grid);
	hipMalloc((void **) &grid_gpu, size);
	hipMalloc((void **) &new_grid_gpu, size);
	hipMemcpy(&grid_gpu, global, size, hipMemcpyHostToDevice);

	/* Kernel invocation */
	dim3 dimBlock(16, 16);
	dim3 dimGrid;
	dimGrid.x = (dimension_x + dimBlock.x - 1) / dimBlock.x;
	dimGrid.y = (dimension_y + dimBlock.y - 1) / dimBlock.y;
	double start, end; 
	j=0;
	GET_TIME(start);
	while(stop != 0  &&  j < 100){

		Calcutalate_AllCells_Kernel<<<dimGrid, dimBlock>>>(&grid_gpu, &new_grid_gpu, dimension_x, dimension_y, filter);  
		if (hipGetLastError() != hipSuccess) {
			printf("Kernel launch failed\n");
		}


		hipMemcpy(&new_grid, &new_grid_gpu, size, hipMemcpyDeviceToHost);



		/*check if there is changes in the grids -- Every process check if there was any changes 
		 *in its grid and sends (throught reduce) 0 if there was not or 1 if there was
		 *-- reduce sums up all the values broadcast it to anybody 
		 *-- if everyone sent 0 the broadcasting sum will be 0 so that means that no
		 *changes took place in any process's grid so the convolution has ended */
		if( (j+1)%10 == 0){
			int result = isEqual_Grid(&new_grid, global, dimension_x, dimension_y); 
			if(result)	stop = -1; //terminate.
		}
		//swaping new with old grids				
		float **temp = new_grid;
		new_grid = (*global);
		(*global) = temp;
		j++;

		hipMemcpy(&grid_gpu, global, size, hipMemcpyHostToDevice);
	}
	GET_TIME(end);
	(*rep) = j;
	hipFree(grid_gpu);
	hipFree(new_grid_gpu);
    Destroy_Grid(&new_grid);
    return end - start ;
}





static inline void Calcutalate_AllCells_(float ***grid, float ***new_grid, int dimension_x, int dimension_y, float filter[FILTER_SIZE][FILTER_SIZE]){
	
	/* The variables below are used to iterate the grid */
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;

	if(i > dimension_x || j > dimension_y) 	return;
	if(i==0 && j==0){ //1) panw aristera gwnia
		((*new_grid)[i])[j]=  
		((*grid)[i+1])[j+1]*filter[0][0]+ ((*grid)[i+1])[j]*filter[0][1]+ ((*grid)[i])[j]*filter[0][2]+
		((*grid)[i])[j+1]*filter[1][0]+ ((*grid)[i])[j]*filter[1][1]+ ((*grid)[i])[j]*filter[1][2]+
		((*grid)[i])[j]*filter[2][0]+ ((*grid)[i])[j]*filter[2][1]+ ((*grid)[i])[j]*filter[2][2];
	}
	else if(i==dimension_x-1 && j==0){ //2) panw de3ia gwnia
		((*new_grid)[i])[j]=  
		((*grid)[i])[j]*filter[0][0]+ ((*grid)[i])[j]*filter[0][1]+ ((*grid)[i])[j]*filter[0][2]+
		((*grid)[i])[j+1]*filter[1][0]+ ((*grid)[i])[j]*filter[1][1]+ ((*grid)[i])[j]*filter[1][2]+
		((*grid)[i-1])[j+1]*filter[2][0]+ ((*grid)[i-1])[j]*filter[2][1]+ ((*grid)[i])[j]*filter[2][2];
	}
	else if(i==0 && j==dimension_y-1){ //3) katw aristera gwnia
		((*new_grid)[i])[j]=  
		((*grid)[i])[j]*filter[0][0]+ ((*grid)[i+1])[j]*filter[0][1]+ ((*grid)[i+1])[j-1]*filter[0][2]+
		((*grid)[i])[j]*filter[1][0]+ ((*grid)[i])[j]*filter[1][1]+ ((*grid)[i])[j-1]*filter[1][2]+
		((*grid)[i])[j]*filter[2][0]+ ((*grid)[i])[j]*filter[2][1]+ ((*grid)[i])[j]*filter[2][2]; 
	}
	else if(i==dimension_x-1 && j=dimension_y-1){ //4) katw de3ia gwnia
		((*new_grid)[i])[j]=  
		((*grid)[i])[j]*filter[0][0]+ ((*grid)[i])[j]*filter[0][1]+ ((*grid)[i])[j]*filter[0][2]+
		((*grid)[i])[j]*filter[1][0]+ ((*grid)[i])[j]*filter[1][1]+ ((*grid)[i])[j-1]*filter[1][2]+
		((*grid)[i])[j]*filter[2][0]+ ((*grid)[i-1])[j]*filter[2][1]+ ((*grid)[i-1])[j-1]*filter[2][2]; 
	}
	else if(i==0){ // 5)aristeri pleura
		((*new_grid)[i])[j]=  
		((*grid)[i+1])[j+1]*filter[0][0]+ ((*grid)[i+1])[j]*filter[0][1]+ ((*grid)[i+1])[j-1]*filter[0][2]+
		((*grid)[i])[j+1]*filter[1][0]+ ((*grid)[i])[j]*filter[1][1]+ ((*grid)[i])[j-1]*filter[1][2]+
		((*grid)[i])[j]*filter[2][0]+ ((*grid)[i])[j]*filter[2][1]+ ((*grid)[i])[j]*filter[2][2];
	}
	else if(j==0){ // 6)panw pleura
		((*new_grid)[i])[j]=  
		((*grid)[i+1])[j+1]*filter[0][0]+ ((*grid)[i+1])[j]*filter[0][1]+ ((*grid)[i])[j]*filter[0][2]+
		((*grid)[i])[j+1]*filter[1][0]+ ((*grid)[i])[j]*filter[1][1]+ ((*grid)[i])[j]*filter[1][2]+
		((*grid)[i-1])[j+1]*filter[2][0]+ ((*grid)[i-1])[j]*filter[2][1]+ ((*grid)[i])[j]*filter[2][2]; 
	}
	else if(i==dimension_x-1){ //7) katw pleura
		((*new_grid)[i])[j]=  
		((*grid)[i])[j]*filter[0][0]+ ((*grid)[i+1])[j]*filter[0][1]+ ((*grid)[i+1])[j-1]*filter[0][2]+
		((*grid)[i])[j]*filter[1][0]+ ((*grid)[i])[j]*filter[1][1]+ ((*grid)[i])[j-1]*filter[1][2]+
		((*grid)[i])[j]*filter[2][0]+ ((*grid)[i-1])[j]*filter[2][1]+ ((*grid)[i-1])[j-1]*filter[2][2]; 
	}
	else if(j==dimension_y-1){
		((*new_grid)[i])[j]=  
		((*grid)[i])[j]*filter[0][0]+ ((*grid)[i])[j]*filter[0][1]+ ((*grid)[i])[j]*filter[0][2]+
		((*grid)[i])[j+1]*filter[1][0]+ ((*grid)[i])[j]*filter[1][1]+ ((*grid)[i])[j-1]*filter[1][2]+
		((*grid)[i-1])[j+1]*filter[2][0]+ ((*grid)[i-1])[j]*filter[2][1]+ ((*grid)[i-1])[j-1]*filter[2][2]; 
	}
	else{ // ola ta alla
		((*new_grid)[i])[j]=  
		((*grid)[i+1])[j+1]*filter[0][0]+ ((*grid)[i+1])[j]*filter[0][1]+ ((*grid)[i+1])[j-1]*filter[0][2]+
		((*grid)[i])[j+1]*filter[1][0]+ ((*grid)[i])[j]*filter[1][1]+ ((*grid)[i])[j-1]*filter[1][2]+
		((*grid)[i-1])[j+1]*filter[2][0]+ ((*grid)[i-1])[j]*filter[2][1]+ ((*grid)[i-1])[j-1]*filter[2][2]; 
	}
}